#include "hip/hip_runtime.h"
//
// Created by jiashuai on 17-9-20.
//
#include <thundersvm/syncarray.h>
#include <hipsparse.h>
#include "thundersvm/kernel/kernelmatrix_kernel.h"
#include <thundersvm/config.h>

#include <hipblas.h>
#include <hipcub/hipcub.hpp>
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;
#define TDEF(x_) std::chrono::high_resolution_clock::time_point x_##_t0, x_##_t1;
#define TSTART(x_) x_##_t0 = Clock::now();
#define TEND(x_) x_##_t1 = Clock::now();
#define TPRINT(x_, str) printf("%-20s \t%.6f\t sec\n", str, std::chrono::duration_cast<std::chrono::microseconds>(x_##_t1 - x_##_t0).count()/1e6);
#define TINT(x_) std::chrono::duration_cast<std::chrono::microseconds>(x_##_t1 - x_##_t0).count()
extern long long time1;
extern long long time3;

using namespace hipcub;
namespace svm_kernel {
    
    __global__ void
    kernel_get_working_set_ins(const kernel_type *val, const int *col_ind, const int *row_ptr, const int *data_row_idx,
                               kernel_type *data_rows,
                               int m, int n) {
        KERNEL_LOOP(i, m) {
            int row = data_row_idx[i];
            for (int j = row_ptr[row]; j < row_ptr[row + 1]; ++j) {
                int col = col_ind[j];
                data_rows[col * m + i] = val[j]; // col-major for cuSPARSE
            }

        }
    }


    __global__ void
    kernel_get_working_set_ins_dns(const kernel_type *val, const int *data_row_idx,
                               kernel_type *data_rows,
                               int m, int n,int n_instances) {
        
        KERNEL_LOOP(i, m) {
            int row = data_row_idx[i];
            for (int j = 0; j < n; ++j) {

                //data_rows[i*n + j] = val[row*n+j]; // row-major for cublas

                // data_rows[i + j*m] = val[row*n+j]; // col-major for cublas

                data_rows[i + j*m] = val[row+j*n_instances]; // col-major for cublas, val col major

            }
        }
            
    }

    __global__ void
    kernel_RBF_kernel(const kernel_type *self_dot0, const kernel_type *self_dot1, kernel_type *dot_product, int m, int n,
                      kernel_type gamma) {
        //m rows of kernel matrix, where m is the working set size; n is the number of training instances
        KERNEL_LOOP(idx, m * n) {
            int i = idx / n;//i is row id
            int j = idx % n;//j is column id
            dot_product[idx] = expf(-(self_dot0[i] + self_dot1[j] - dot_product[idx] * 2) * gamma);
        }
    }

    __global__ void
    kernel_RBF_kernel(const int *self_dot0_idx, const kernel_type *self_dot1, kernel_type *dot_product, int m, int n,
                      kernel_type gamma) {
        //compute m rows of kernel matrix, where m is the working set size and n is the number of training instances, according to idx
        KERNEL_LOOP(idx, m * n) {
            int i = idx / n;//i is row id
            int j = idx % n;//j is column id
            dot_product[idx] = expf(-(self_dot1[self_dot0_idx[i]] + self_dot1[j] - dot_product[idx] * 2) * gamma);
        }
    }

    __global__ void
    kernel_sum_kernel_values(const float_type *coef, int total_sv, const int *sv_start, const int *sv_count,
                             const float_type *rho,
                             const kernel_type *k_mat, float_type *dec_values, int n_classes, int n_instances) {
        KERNEL_LOOP(idx, n_instances) {
            int k = 0;
            int n_binary_models = n_classes * (n_classes - 1) / 2;
            for (int i = 0; i < n_classes; ++i) {
                for (int j = i + 1; j < n_classes; ++j) {
                    int si = sv_start[i];
                    int sj = sv_start[j];
                    int ci = sv_count[i];
                    int cj = sv_count[j];
                    const float_type *coef1 = &coef[(j - 1) * total_sv];
                    const float_type *coef2 = &coef[i * total_sv];
                    const kernel_type *k_values = &k_mat[idx * total_sv];
                    double sum = 0;
                    for (int l = 0; l < ci; ++l) {
                        sum += coef1[si + l] * k_values[si + l];
                    }
                    for (int l = 0; l < cj; ++l) {
                        sum += coef2[sj + l] * k_values[sj + l];
                    }
                    dec_values[idx * n_binary_models + k] = sum - rho[k];
                    k++;
                }
            }
        }
    }

    __global__ void
    kernel_poly_kernel(kernel_type *dot_product, kernel_type gamma, kernel_type coef0, int degree, int mn) {
        KERNEL_LOOP(idx, mn) {
            dot_product[idx] = powf(gamma * dot_product[idx] + coef0, degree);
        }
    }

    __global__ void kernel_sigmoid_kernel(kernel_type *dot_product, kernel_type gamma, kernel_type coef0, int mn) {
        KERNEL_LOOP(idx, mn) {
            dot_product[idx] = tanhf(gamma * dot_product[idx] + coef0);
        }
    }

    void sum_kernel_values(const SyncArray<float_type> &coef, int total_sv, const SyncArray<int> &sv_start,
                           const SyncArray<int> &sv_count, const SyncArray<float_type> &rho,
                           const SyncArray<kernel_type> &k_mat,
                           SyncArray<float_type> &dec_values, int n_classes, int n_instances) {
        SAFE_KERNEL_LAUNCH(kernel_sum_kernel_values, coef.device_data(), total_sv, sv_start.device_data(),
                           sv_count.device_data(), rho.device_data(), k_mat.device_data(), dec_values.device_data(),
                           n_classes, n_instances);

    }

    void
    get_working_set_ins(const SyncArray<kernel_type> &val, const SyncArray<int> &col_ind, const SyncArray<int> &row_ptr,
                        const SyncArray<int> &data_row_idx, SyncArray<kernel_type> &data_rows, int m, int n) {
        SAFE_KERNEL_LAUNCH(kernel_get_working_set_ins, val.device_data(), col_ind.device_data(), row_ptr.device_data(),
                           data_row_idx.device_data(), data_rows.device_data(), m, n);



    }

    void
    get_working_set_ins_dns(const SyncArray<kernel_type> &val, 
                            const SyncArray<int> &data_row_idx, SyncArray<kernel_type> &data_rows, int m, int n,int n_instances){

        SAFE_KERNEL_LAUNCH(kernel_get_working_set_ins_dns, val.device_data(),
                           data_row_idx.device_data(), data_rows.device_data(), m, n,n_instances);
    }

    void
    RBF_kernel(const SyncArray<kernel_type> &self_dot0, const SyncArray<kernel_type> &self_dot1,
               SyncArray<kernel_type> &dot_product, int m,
               int n,
               kernel_type gamma) {
        SAFE_KERNEL_LAUNCH(kernel_RBF_kernel, self_dot0.device_data(), self_dot1.device_data(),
                           dot_product.device_data(), m, n, gamma);
    }

    void
    RBF_kernel(const SyncArray<int> &self_dot0_idx, const SyncArray<kernel_type> &self_dot1,
               SyncArray<kernel_type> &dot_product, int m,
               int n, kernel_type gamma) {
        SAFE_KERNEL_LAUNCH(kernel_RBF_kernel, self_dot0_idx.device_data(), self_dot1.device_data(),
                           dot_product.device_data(), m, n, gamma);
    }

    void poly_kernel(SyncArray<kernel_type> &dot_product, kernel_type gamma, kernel_type coef0, int degree, int mn) {
        SAFE_KERNEL_LAUNCH(kernel_poly_kernel, dot_product.device_data(), gamma, coef0, degree, mn);
    }

    void sigmoid_kernel(SyncArray<kernel_type> &dot_product, kernel_type gamma, kernel_type coef0, int mn) {
        SAFE_KERNEL_LAUNCH(kernel_sigmoid_kernel, dot_product.device_data(), gamma, coef0, mn);
    }

    hipsparseHandle_t handle;
    hipsparseMatDescr_t descr;
    bool cusparse_init;
    hipblasHandle_t handle2;
    //m for instance; n for get_rows num; k for feature num; nnz for number of nonzero
    void dns_csr_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &csr_val,
                     const SyncArray<int> &csr_row_ptr, const SyncArray<int> &csr_col_ind, int nnz,
                     SyncArray<kernel_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;

            hipblasCreate(&handle2);
        }
        kernel_type one(1);
        kernel_type zero(0);

#if (CUDART_VERSION >= 11000)

        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;
        
#ifdef USE_DOUBLE
        hipDataType data_type = HIP_R_64F;
#else//kernel type is float
        hipDataType data_type = HIP_R_32F;
#endif  

        hipsparseCreateCsr(&matA, m, k, nnz, (void*)csr_row_ptr.device_data(), (void*)csr_col_ind.device_data(),
                          (void*)csr_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, data_type);
        //hipsparseCreateDnMat(&matB, n, k, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_COL);
        //hipsparseCreateDnMat(&matC, m, n, m, (void*)result.device_data(), data_type, HIPSPARSE_ORDER_COL);
        //
       
        //
        //size_t buffer_size = 0;
        //hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        //                       &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1,
        //                       &buffer_size);

        //void *p_buffer = nullptr;
        //
        //hipMalloc((void**)&p_buffer, buffer_size);
        //
        //hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        //           &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);
        //hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
        //           &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);
        
        
        hipsparseCreateDnMat(&matB, k, n, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_ROW);
        SyncArray<kernel_type> tmp_res(m*n);
        hipsparseCreateDnMat(&matC, m, n, n, (void*)tmp_res.device_data(), data_type, HIPSPARSE_ORDER_ROW);

        size_t buffer_size = 0;
        hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG2,
                                &buffer_size);

        void *p_buffer = nullptr;
        hipMalloc((void**)&p_buffer, buffer_size);
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                           &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG2, p_buffer);
        hipblasStatus_t success=hipblasSgeam( handle2, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, 
                                            &one, tmp_res.device_data(), n, &zero, tmp_res.device_data(), m, 
                                            result.device_data(), m);


        hipFree(p_buffer);
        
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);

        hipDeviceSynchronize();    

#else

#ifdef USE_DOUBLE
        hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                        m, n, k, nnz, &one, descr, csr_val.device_data(), csr_row_ptr.device_data(),
                        csr_col_ind.device_data(),
                        dense_mat.device_data(), n, &zero, result.device_data(), m);
#else//kernel type is float
        hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                        m, n, k, nnz, &one, descr, csr_val.device_data(), csr_row_ptr.device_data(),
                        csr_col_ind.device_data(),
                        dense_mat.device_data(), n, &zero, result.device_data(), m);

        //hipsparseScsrmm return row-major matrix, so no transpose is needed
#endif // ifdef USE_DOUBLE

#endif // if CUDART_VERSION >= 11000
    }



    //dns dns mul

    hipblasHandle_t handle_blas;
    bool cublas_init;
    void dns_dns_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_a,const SyncArray<kernel_type> &dense_b,kernel_type beta, 
                     SyncArray<kernel_type> &result){

        if (!cublas_init) {
            hipblasCreate(&handle_blas);
            cublas_init = true;
        }

        kernel_type alpha=1.0;
        const kernel_type* d_dense_a = dense_a.device_data();
        const kernel_type* d_dense_b = dense_b.device_data();

        // hipblasSgemm(handle_blas,HIPBLAS_OP_T,HIPBLAS_OP_N, m, n, k,&alpha,dense_a.device_data(), k, dense_b.device_data(), k,&beta, result.device_data(), m);

        //dense b :k*n
        // hipblasSgemm(handle_blas,HIPBLAS_OP_T,HIPBLAS_OP_T, m, n, k,&alpha,dense_a.device_data(), k, dense_b.device_data(), n,&beta, result.device_data(), m);
        hipblasSgemm(handle_blas,HIPBLAS_OP_N,HIPBLAS_OP_T, m, n, k,&alpha,dense_a.device_data(), m, dense_b.device_data(), n,&beta, result.device_data(), m);
        

    }

    //csr csr mul
    void csr_csr_mul_cuda(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &csr_val,
                     const SyncArray<int> &csr_row_ptr, const SyncArray<int> &csr_col_ind, int nnz,
                     SyncArray<kernel_type> &result){

        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }

        
        kernel_type alpha(1);
        kernel_type beta(0);
        hipDataType data_type = HIP_R_32F;
        hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

        hipsparseSpMatDescr_t matA, matB, matC;
        
        hipsparseDnMatDescr_t tmp_mat,result_mat;

        

        void* dBuffer = NULL;
        size_t bufferSize = 0;

        void*  dBuffer1 = NULL, *dBuffer2 = NULL,*dBuffer3 = NULL;
        size_t bufferSize1 = 0, bufferSize2 = 0 ,bufferSize3= 0;

        

        int *tmp_csr_row;
        hipMalloc((void**) &tmp_csr_row,(k + 1) * sizeof(int));

        //create matrix
        hipsparseCreateCsr(&matA, m, k, nnz, (void*)csr_row_ptr.device_data(), (void*)csr_col_ind.device_data(),
                          (void*)csr_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateCsr(&matB, k, n, 0,
                                      tmp_csr_row, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateCsr(&matC, m, n, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateDnMat(&result_mat, m, n, m, (void*)result.device_data(),
                                        data_type, HIPSPARSE_ORDER_COL);

        
     
        //dense转化为csr格式, shape k*n
        
        hipsparseCreateDnMat(&tmp_mat, k, n, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_ROW);

        hipsparseDenseToSparse_bufferSize(
                                        handle, tmp_mat, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize);
        hipMalloc((void**)&dBuffer, bufferSize);


        hipsparseDenseToSparse_analysis(handle, tmp_mat, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer);

        int64_t num_rows_tmp, num_cols_tmp, nnz_tmp;
        int *d_csr_columns;
        
        kernel_type* d_csr_values;

        hipsparseSpMatGetSize(matB, &num_rows_tmp, &num_cols_tmp,&nnz_tmp);
        
        hipMalloc((void**) &d_csr_columns, nnz_tmp * sizeof(int));
        hipMalloc((void**) &d_csr_values,  nnz_tmp * sizeof(kernel_type));
        

        hipsparseCsrSetPointers(matB, tmp_csr_row, d_csr_columns,d_csr_values);
        hipsparseDenseToSparse_convert(handle, tmp_mat, matB,
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer);


        //timing
        hipEvent_t start_event, stop_event;
        float cuda_elapsed_ms  = 0;
        hipEventCreate(&start_event);
        hipEventCreate(&stop_event);
        hipEventRecord(start_event, NULL);
        //csr csr mul

        hipsparseSpGEMMDescr_t spgemmDesc;
        hipsparseSpGEMM_createDescr(&spgemmDesc);

        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      data_type, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL);
        hipMalloc((void**) &dBuffer1, bufferSize1);

        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      data_type, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1);

        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               data_type, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL);

        hipMalloc((void**) &dBuffer2, bufferSize2);

        hipsparseSpGEMM_compute(handle, opA, opB,
                                           &alpha, matA, matB, &beta, matC,
                                           data_type, HIPSPARSE_SPGEMM_DEFAULT,
                                           spgemmDesc, &bufferSize2, dBuffer2);

        int64_t C_num_rows1, C_num_cols1, C_nnz1;
        hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,&C_nnz1);

        int *dC_csrOffsets,*dC_columns;
        kernel_type* dC_values;

        hipMalloc((void**) &dC_csrOffsets, (m+1) * sizeof(int));
        hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int));
        hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(kernel_type));
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values);
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            data_type, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);

        hipEventRecord(stop_event, NULL);
        hipEventSynchronize(stop_event);
        hipEventElapsedTime(&cuda_elapsed_ms, start_event,stop_event);
        LOG(INFO)<<"csr csr mul time is "<<cuda_elapsed_ms;
        //csr to dns

        hipsparseSparseToDense_bufferSize(
                                        handle, matC, result_mat,
                                        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                        &bufferSize3);
        hipMalloc(&dBuffer3, bufferSize3);
        
        hipsparseSparseToDense(handle, matC, result_mat,
                                          HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
                                          dBuffer3);
        
        

        hipsparseDestroySpMat(matA);
        hipsparseDestroySpMat(matB);
        hipsparseDestroySpMat(matC);
        hipsparseSpGEMM_destroyDescr(spgemmDesc);
        hipsparseDestroyDnMat(result_mat);
        hipFree(dBuffer);
        hipFree(dBuffer1);
        hipFree(dBuffer2);
        hipFree(dBuffer3);
        hipFree(tmp_csr_row);
        hipFree(d_csr_columns);
        hipFree(d_csr_values);
        hipsparseDestroyDnMat(tmp_mat);
        hipFree(dBuffer);
        hipFree(dC_csrOffsets);
        hipFree(dC_columns);
        hipFree(dC_values);




    } 


    //bsr dns mul

    void bsr_dns_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &bsr_val,
                     const SyncArray<int> &bsr_row_ptr, const SyncArray<int> &bsr_col_ind, 
                     SyncArray<kernel_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }
        kernel_type alpha(1);
        kernel_type beta(0);

        hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;

        int nnzb = bsr_col_ind.size();
        int mb = bsr_row_ptr.size()-1;
        int blockSize = sqrt(bsr_val.size()/nnzb);
        int nb = (k+blockSize-1)/blockSize;

        //mul
        hipsparseSbsrmm(handle,
               dir,
               HIPSPARSE_OPERATION_NON_TRANSPOSE,
               HIPSPARSE_OPERATION_TRANSPOSE,
               mb, n, nb, nnzb, &alpha,
               descr, bsr_val.device_data(), bsr_row_ptr.device_data(), bsr_col_ind.device_data(), blockSize,
               dense_mat.device_data(), n,
               &beta, result.device_data(), m);
    }



    void csc_dns_mul(int m, int n, int k, const SyncArray<kernel_type> &dense_mat, const SyncArray<kernel_type> &csc_val,
                     const SyncArray<int> &csc_row_ptr, const SyncArray<int> &csc_col_ind, int nnz,
                     SyncArray<kernel_type> &result) {
        if (!cusparse_init) {
            hipsparseCreate(&handle);
            hipsparseCreateMatDescr(&descr);
            hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
            hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
            cusparse_init = true;
        }
        
        kernel_type one(1);
        kernel_type zero(0);

        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;

        hipDataType data_type = HIP_R_32F;

        // hipsparseCreateCsr(&matA, k, m, nnz, (void*)csc_col_ind.device_data(), (void*)csc_row_ptr.device_data(),
        //                   (void*)csc_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        //                   HIPSPARSE_INDEX_BASE_ZERO, data_type);

        hipsparseCreateCsc(&matA, m, k, nnz, (void*)csc_col_ind.device_data(), (void*)csc_row_ptr.device_data(),
                          (void*)csc_val.device_data(), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, data_type);
        hipsparseCreateDnMat(&matB, n, k, n, (void*)dense_mat.device_data(), data_type, HIPSPARSE_ORDER_COL);
        hipsparseCreateDnMat(&matC, m, n, m, (void*)result.device_data(), data_type, HIPSPARSE_ORDER_COL);

        size_t buffer_size = 0;
        hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                                &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1,
                                &buffer_size);

        void *p_buffer = nullptr;
        hipMalloc((void**)&p_buffer, buffer_size);

        hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                   &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);
        hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE,
                    &one, matA, matB, &zero, matC, data_type, HIPSPARSE_SPMM_CSR_ALG1, p_buffer);

        hipFree(p_buffer);
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);
    }

}


